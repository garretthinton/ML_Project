#include "hip/hip_runtime.h"
//	University of Arizona
//	Center for Gamma-Ray Imaging
//	
//
//
//
//
//
//
//  
// Author and Revision Date 
// Garrett Hinton		Jun 13 2017
//
//
//
// Title:	****Contracting_Grid****
// Description:  This class takes in a frame, as well as some parameters for how the user would like the algorithm to behave.  // It has the capabilities to output the center points for the first and second screen in the imaging process.  It also has the 
// ability to output the direction of propagation for beta particle being imaged.  The idea of this class is that using the
// Cine class, the user could read in the data and analyze it in real time.
		

#include "Contracting_Grid.h"

using namespace std;
		
#define MAP_2D(__dimx, __dimy, __x, __y)		((__y) * (__dimx) + (__x))

#define CHECK(x) do {\
	hipError_t err = (x);\
	if ( err!= hipSuccess)\
	{\
		fprintf(stderr, "API error %s:%d Returned:%d\n", \
		__FILE__, __LINE__, err);\
		printf(hipGetErrorString(err));\
		exit(1);\
	}\
	}while(0)
		
// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid()
{
	// standard assumptions for algorithms
	iterations = 7;
	shrinkFactor = 2;
	distance = .075;
	
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y  = real_dim_x = real_dim_y = dim_x = dim_y = 0;
	
	// May need to instantiate 'frame'
	frame = NULL;
}

// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid(Cine c)
{
	// standard assumptions for algorithms
	iterations = 7;
	shrinkFactor = 2;
	distance = .075;
	
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y = 0;
	
	// Get dimensions of the frame
	dim_x = c.Dim_x();
	dim_y = c.Dim_y();
	
	unsigned int dim_Max = dim_x;
	
	if(dim_y > dim_x)
	{
		dim_Max = dim_y;
	}
	
	iterations = log(dim_Max)/log(shrinkFactor);
	
	real_dim_x = dim_x + (dim_x / shrinkFactor);
	real_dim_y = dim_y + (dim_y / shrinkFactor);
}

// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid(Cine c, float shrinkFactor_in)
{
	// standard assumptions for algorithms
	//Checks that the value for the shrinkFactor is acceptable
	if(shrinkFactor_in > 1)
	{
		shrinkFactor = shrinkFactor_in;
	}
	
	distance = .075;
	
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y = 0;
	
	// Get dimensions of the frame
	dim_x = c.Dim_x();
	dim_y = c.Dim_y();
	
	unsigned int dim_Max = dim_x;
	
	if(dim_y > dim_x)
	{
		dim_Max = dim_y;
	}
	
	iterations = log(dim_Max)/log(shrinkFactor);
	
	real_dim_x = dim_x + (dim_x / shrinkFactor);
	real_dim_y = dim_y + (dim_y / shrinkFactor);
}

// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid(Cine c, unsigned int iterations_in, float shrinkFactor_in)
{
	// standard assumptions for algorithms			
	distance = .075;
	
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y = 0;
	
	// Get dimensions of the frame
	dim_x = c.Dim_x();
	dim_y = c.Dim_y();
	
	//Checks that the value for iterations is acceptable
	if(iterations_in > 0)
	{
		iterations = iterations_in;
	}
	
	//Checks that the value for the shrinkFactor is acceptable
	if(shrinkFactor_in > 1)
	{
		shrinkFactor = shrinkFactor_in;
	}
	
	real_dim_x = dim_x + (dim_x / shrinkFactor);
	real_dim_y = dim_y + (dim_y / shrinkFactor);
}

// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid(Cine c, float shrinkFactor_in, float distance_in)
{
	shrinkFactor = 2;
	distance = .075;
	
	if(shrinkFactor_in > 1)
	{
		shrinkFactor = shrinkFactor_in;
	}
	
	if(distance_in > 0)
	{
		distance = distance_in;
	}
	
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y = 0;
	
	// Get dimensions of the frame
	dim_x = c.Dim_x();
	dim_y = c.Dim_y();
	
	unsigned int dim_Max = dim_x;
	
	if(dim_y > dim_x)
	{
		dim_Max = dim_y;
	}
	
	iterations = log(dim_Max)/log(shrinkFactor);
	
	real_dim_x = dim_x + (dim_x / shrinkFactor);
	real_dim_y = dim_y + (dim_y / shrinkFactor);
}

// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid(Cine c, unsigned int iterations_in, float shrinkFactor_in, float distance_in)
{
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y = 0;
	
	// Get dimensions of the frame
	dim_x = c.Dim_x();
	dim_y = c.Dim_y();
	
	//Checks that the value for iterations is acceptable
	if(iterations_in > 0)
	{
		iterations = iterations_in;
	}
	
	//Checks that the value for the shrinkFactor is acceptable
	if(shrinkFactor_in > 1)
	{
		shrinkFactor = shrinkFactor_in;
	}

	//Checks that the value for the distance is acceptable
	if(distance_in > 0)
	{
		distance = distance_in;
	}
	
	real_dim_x = dim_x + (dim_x / shrinkFactor);
	real_dim_y = dim_y + (dim_y / shrinkFactor);
}


// referred to in Contracting_Grid.h
Contracting_Grid::Contracting_Grid(unsigned int iterations_in, float shrinkFactor_in, float distance_in)
{
	// Set these numbers to 0.
	sharp_x = sharp_y = broad_x = broad_y = 0;
	direction_x = direction_y = real_dim_x = real_dim_y = 0;
	
	// The real dimensions of the frame will be received later if this is the contructor used
	dim_x = dim_y = 0;
	
	frame = NULL;
	//Checks that the value for iterations is acceptable
	if(iterations_in > 0)
	{
		iterations = iterations_in;
	}
	
	//Checks that the value for the shrinkFactor is acceptable
	if(shrinkFactor_in > 1)
	{
		shrinkFactor = shrinkFactor_in;
	}

	//No check is needed yet
	distance = distance_in;	
}

 // referred to in Contracting_Grid.h
/* void Contracting_Grid::getFrameData(&Cine c)
{
	// Get the x and y dimensions to know how much memory to allocate for the frame
	dim_x = c.Dim_x();
	dim_y = c.Dim_y();
	
	real_dim_x = dim_x + (dim_x / shrinkFactor);
	real_dim_y = dim_y + (dim_y / shrinkFactor);
	
	// Allocate memory 
	//frame = new float[dim_x * dim_y];
	malloc(frame, real_dim_x * real_dim_y * sizeof(float));
	// Copy the values over
	for(i = 0; i < dim_x; ++i) {
	  for(j = 0; j < dim_y; ++j) {
		//frame[MAP_2D(dim_x, dim_y, i, j,)] = c.data_frame[MAP_2D(dim_x, dim_y, i, j)];
		frame[MAP_2D(dim_x, dim_y, dim_x/(2*shrinkFactor) + i, dim_y/(2*shrinkFactor) + j)] = c.data_frame[MAP_2D(dim_x, dim_y, i, j)];
	  }
	}
}  */

// Essence of the contracting grid algorithm and the purpose of this class.  Referred to in Contracting_Grid.h
void Contracting_Grid::findSharpCenter_Rec()
{
	float *dev_frame_long;
	
	// This is the frame data that will be going into the GPU
	float *dev_frame;
	unsigned int *max;
	unsigned int *dev_center_x;
	unsigned int *dev_center_y;
	unsigned int iteration_curr = 0;
	unsigned int center_x = real_dim_x / 2;
	unsigned int center_y = real_dim_y / 2;
	unsigned int totalSamples = real_dim_x * real_dim_y;
	unsigned int dim_Samples = dim_x * dim_y;
	
	// Allocate memory in the GPU
	CHECK(hipMalloc((void**)&dev_frame, dim_Samples * sizeof(float)));
	CHECK(hipMalloc((float**)&dev_frame_long, totalSamples * sizeof(float)));
	
	CHECK(hipMalloc((void**)&max, sizeof(unsigned int)));
	CHECK(hipMalloc((void**)&dev_center_x, sizeof(unsigned int)));
	CHECK(hipMalloc((void**)&dev_center_y, sizeof(unsigned int)));
	
	// Copy the data to the GPU
	CHECK(hipMemcpy(dev_frame, frame, dim_Samples * sizeof(float), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_center_x, &center_x, sizeof(unsigned int), hipMemcpyHostToDevice));
	CHECK(hipMemcpy(dev_center_y, &center_y, sizeof(unsigned int), hipMemcpyHostToDevice));
	
	g_Zero_Array<<<real_dim_x, real_dim_y>>>(	dev_frame,
												dev_frame_long,
												dim_x,
												dim_y);
	
	g_Initialize_Array<<<dim_x, dim_y>>>(	dev_frame,
											dev_frame_long,
											real_dim_x,
											real_dim_y);
	
	// may need a hipDeviceSynchronize here
	hipDeviceSynchronize();
	CHECK(hipGetLastError());
	
	g_Contracting_Max<<<1,16>>>(dev_frame_long,
								max, 									
								dev_center_x,
								dev_center_y,
								real_dim_x,
								real_dim_y,
								iterations, 
								iteration_curr,
								shrinkFactor);
	
	CHECK(hipGetLastError());
	// may need a hipDeviceSynchronize here
	hipDeviceSynchronize();
	
	// Copy the data back to the host
	CHECK(hipMemcpy(&center_x, (void*)dev_center_x, sizeof(unsigned int), hipMemcpyDeviceToHost));
	CHECK(hipMemcpy(&center_y, (void*)dev_center_y, sizeof(unsigned int), hipMemcpyDeviceToHost));
	
	sharp_x = center_x - ((real_dim_x - dim_x) / 2);
	sharp_y = center_y - ((real_dim_y - dim_y) / 2);
	
	
	/*
	cout<< "sharp_x: " << sharp_x << endl;
	cout<< "sharp_y: " << sharp_y << endl;
	
	cout<< "frame[sharp_x, sharp_y]: "		<<  frame[MAP_2D(dim_x,dim_y, sharp_x, sharp_y)]<<endl;
	cout<< "frame[sharp_x-1, sharp_y]: "	<<    frame[MAP_2D(dim_x,dim_y,sharp_x - 1, sharp_y)]<<endl;
	cout<< "frame[sharp_x+1, sharp_y]: "	<<    frame[MAP_2D(dim_x,dim_y,sharp_x + 1, sharp_y)]<<endl;
	cout<< "frame[sharp_x, sharp_y-1]: "	<<   frame[MAP_2D(dim_x,dim_y,sharp_x, sharp_y - 1)]<<endl;
	cout<< "frame[sharp_x-1, sharp_y-1]: "	<<   frame[MAP_2D(dim_x,dim_y,sharp_x - 1, sharp_y - 1)]<<endl;
	cout<< "frame[sharp_x+1, sharp_y-1]: "	<<    frame[MAP_2D(dim_x,dim_y,sharp_x + 1, sharp_y - 1)]<<endl;
	cout<< "frame[sharp_x, sharp_y+1]: "	<<   frame[MAP_2D(dim_x,dim_y,sharp_x, sharp_y + 1)]<<endl;
	cout<< "frame[sharp_x-1, sharp_y+1]: "	<<    frame[MAP_2D(dim_x,dim_y,sharp_x - 1, sharp_y + 1)]<<endl;
	cout<< "frame[sharp_x+1, sharp_y+1]: "	<<   frame[MAP_2D(dim_x,dim_y,sharp_x + 1, sharp_y + 1)]<<endl; 
	
	
	float temp1 = frame[MAP_2D(dim_x,dim_y,0,0)];
	int tempi = 0;
	int tempj = 0;
	for(int i = 0; i< dim_x;i++){
		for(int j = 0; j < dim_y; j++){
			if(temp1 < frame[MAP_2D(dim_x,dim_y,i,j)]){
				temp1 = frame[MAP_2D(dim_x,dim_y,i,j)];
				tempi = i;
				tempj = j;
			}
		}
	}

	
	cout<<"True max: "<< temp1 <<endl;
	cout<< "i , j: \t" << tempi << " , " << tempj <<endl;
	*/
	
	// Free the data
	CHECK(hipFree(dev_frame));
	CHECK(hipFree(dev_frame_long));
	CHECK(hipFree(max));
	CHECK(hipFree(dev_center_x));
	CHECK(hipFree(dev_center_y));	
}

// referred to in Contracting_Grid.h
void Contracting_Grid::findBroadCenter()
{	
/*
	// This is the frame data that will be going into the GPU
	float *dev_frame;
	unsigned int *max;
	unsigned int *dev_x;
	unsigned int *dev_y;
	unsigned int dim_Samples = dim_x * dim_y;
	
	// Allocate memory in the GPU
	CHECK(hipMalloc((void**)&dev_frame, dim_Samples * sizeof(float)));
	
	// Copy the data to the GPU
	CHECK(hipMemcpy(dev_frame, frame, dim_Samples * sizeof(float), hipMemcpyHostToDevice));
	
	// possibly take the sharpCenter(x,y) and delete those rows and columns + or - 10 lines
	
	//0 or 1 depending on x or y dimension collection
	g_GetValues(dev_frame, 0, );
	g_GetValues(dev_frame, 1, );
	
	// Add up the sums for every 21 lines of data up to the sharpCenter +- 10.  If the lines are increasing, interpolate the data to adjust the data for the lines that were ignored.  Choose the center of the highest 21 lines 
*/
}

// referred to in Contracting_Grid.h
void Contracting_Grid::findDirection()
{
	
}

// Cine Contracting_Grid::operator=(const Cine &c)
// {
	// Cine cine();
	// cine.Filename(c.Filename());
	// cine.Dim_x(c.Dim_x());
	// cine.Dim_y(c.Dim_y());
	// cine.Dim_z(c.Dim_z());
	// cine.read_cine_3d();
// }

//Setter for the frame field
void Contracting_Grid::Frame(float *input_Frame)
{
	frame = new float[dim_x * dim_y];

	for(int i = 0; i<dim_x;i++)
	{
		for(int j = 0; j<dim_y;j++)
		{
			frame[MAP_2D(dim_x, dim_y, i, j)] =  input_Frame[MAP_2D(dim_x, dim_y, i, j)];
		}
	}
}